/**
 * @file
 * Implementation of batched 2D FFTs with CUDA
 *
 * @author Jim Hardwick
 * @date 12 Jan 2008
 *
 * @note
 * This code was posted to nVidia's public CUDA forum as thanks to the CUDA
 * forum membership for their assistance. No copyright was asserted and no
 * license terms were imposed in the post. The post can be found at
 * http://forums.nvidia.com/index.php?showtopic=34241
 */

#include <cutil_inline.h>
#include <assert.h>

#include "batchfft.h"
#include "transpose.cuh"

////////////////////////////////////////////////////////////////////////////////
hipfftResult batchfftFillPlan(batchfftHandle *plan, int nx, int ny, int nz, hipfftType type, int batch)
{
	if(type != HIPFFT_C2C)
		return HIPFFT_INVALID_TYPE;

	if(nx % HALF_WARP_SIZE != 0)
		return HIPFFT_INVALID_SIZE;

	if(ny % HALF_WARP_SIZE != 0)
		return HIPFFT_INVALID_SIZE;

	if(nz != 1 && nz % HALF_WARP_SIZE != 0)
		return HIPFFT_INVALID_SIZE;

	if(nz == 1)
	{
		plan->dim = 2;

		// Swap nx and ny so they correspoind to the 2D CUFFT API.
		// 2D cufft expects them in the order for a declared C array:
		//
		// hipfftComplex array[nx][ny];
		// hipfftPlan2d(plan, nx, ny, type);
		//
		// even though ny would be considered the "x" array index for row-major
		// array storage.
		plan->ny = nx;
		plan->nx = ny;
		plan->nz = 1;
	}
	else
	{
		plan->dim = 3;

		// Swap dimensions, the reason is the same as for 2D case.
		plan->nx = nz;
		plan->ny = ny;
		plan->nz = nx;
	}

	plan->type = type;
	plan->batch = batch;

	hipfftResult ret = HIPFFT_SUCCESS;
	hipError_t cudaret = hipSuccess;

	cudaret = hipMalloc(&(plan->temp), plan->nx * plan->ny * plan->nz * plan->batch * sizeof(hipfftComplex));
	if(cudaret != hipSuccess)
		return HIPFFT_ALLOC_FAILED;

	ret = hipfftPlan1d(&(plan->xplan), plan->nx, plan->type, plan->ny * plan->nz * plan->batch);
	if(ret != HIPFFT_SUCCESS)
	{
		hipFree(plan->temp);
		plan->temp = NULL;
		return ret;
	}

	ret = hipfftPlan1d(&(plan->yplan), plan->ny, plan->type, plan->nx * plan->nz * plan->batch);
	if(ret != HIPFFT_SUCCESS)
	{
		hipFree(plan->temp);
		plan->temp = NULL;
		hipfftDestroy(plan->xplan);
		return ret;
	}

	if(plan->dim == 3)
	{
		ret = hipfftPlan1d(&(plan->zplan), plan->nz, plan->type, plan->nx * plan->ny * plan->batch);
		if(ret != HIPFFT_SUCCESS)
		{
			hipFree(plan->temp);
			plan->temp = NULL;
			hipfftDestroy(plan->xplan);
			hipfftDestroy(plan->yplan);
			return ret;
		}
	}

	return HIPFFT_SUCCESS;
}

hipfftResult batchfftPlan2d(batchfftHandle* plan, int nx, int ny, hipfftType type, int batch)
{
	return batchfftFillPlan(plan, nx, ny, 1, type, batch);
}

hipfftResult batchfftPlan3d(batchfftHandle* plan, int nx, int ny, int nz, hipfftType type, int batch)
{
	return batchfftFillPlan(plan, nx, ny, nz, type, batch);
}

////////////////////////////////////////////////////////////////////////////////
hipfftResult batchfftDestroy(batchfftHandle &plan)
{
	assert(plan.temp != NULL);

	hipfftDestroy(plan.xplan);
	hipfftDestroy(plan.yplan);
	if(plan.dim == 3)
		hipfftDestroy(plan.zplan);
	hipFree(plan.temp);
	plan.temp = NULL;

	return HIPFFT_SUCCESS;
}

hipfftResult batchfftExecute2D(batchfftHandle &plan, hipfftComplex* idata, hipfftComplex* odata, int sign)
{
	hipfftResult cufftret = HIPFFT_SUCCESS;
	hipError_t cudaret = hipSuccess;

	// Transform rows
	cufftret = hipfftExecC2C(plan.xplan, idata, odata, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	// Transpose
	cudaret = transpose(plan.temp, odata, plan.nx, plan.ny, plan.batch * plan.nz);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	// Transform columns
	cufftret = hipfftExecC2C(plan.yplan, plan.temp, plan.temp, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	// Transpose back
	cudaret = transpose(odata, plan.temp, plan.ny, plan.nx, plan.batch * plan.nz);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	return HIPFFT_SUCCESS;
}

hipfftResult batchfftExecute3D(batchfftHandle &plan, hipfftComplex* idata, hipfftComplex* odata, int sign)
{
	hipfftResult cufftret = HIPFFT_SUCCESS;
	hipError_t cudaret = hipSuccess;

	cufftret = batchfftExecute2D(plan, idata, odata, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	cudaret = transpose(plan.temp, odata, plan.nx * plan.ny, plan.nz, plan.batch);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	cufftret = hipfftExecC2C(plan.zplan, plan.temp, plan.temp, sign);
	if(cufftret != HIPFFT_SUCCESS)
		return cufftret;

	cudaret = transpose(odata, plan.temp, plan.nz, plan.nx * plan.ny, plan.batch);
	if(cudaret != hipSuccess)
		return HIPFFT_EXEC_FAILED;

	return HIPFFT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
hipfftResult batchfftExecute(batchfftHandle &plan, hipfftComplex* idata, hipfftComplex* odata, int sign)
{
	if(plan.dim == 2)
		return batchfftExecute2D(plan, idata, odata, sign);
	else if(plan.dim == 3)
		return batchfftExecute3D(plan, idata, odata, sign);
	else
		return HIPFFT_INVALID_PLAN;
}
