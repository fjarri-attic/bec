#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cutil_inline.h>
#include <math.h>

#include "defines.h"
#include "batchfft.h"
#include "reduce.cuh"
#include "spectre.h"
#include "transpose.cuh"
#include "wigner_kernel.cu"
#include "cudatexture.h"

// fill complex vector with normally distributed random numbers
void fillWithNormalDistribution(CudaBuffer<value_pair> &data, value_type dev)
{
	value_pair *h_data = new value_pair[data.len()];

	for(int i = 0; i < data.len(); i++)
	{
		// pair of independent uniformly distributed random numbers
		value_pair num;
		num.x = ((value_type)(rand() + 1))/((unsigned int)RAND_MAX + 1);
		num.y = ((value_type)(rand() + 1))/((unsigned int)RAND_MAX + 1);

		// Box-Muller transform to get two normally distributed random numbers
		h_data[i].x = dev * sqrt(-2.0 * log(num.x)) * cos(2 * M_PI * num.y);
		h_data[i].y = dev * sqrt(-2.0 * log(num.x)) * sin(2 * M_PI * num.y);
	}

	data.copyFrom(h_data);

	delete[] h_data;
}

// returns value of energy/chem.potential integral over given state
// *_temp values are spoiled after call
value_type calculateStateIntegral(CudaBuffer<value_pair> &state,
	CudaBuffer<value_pair> &complex_temp,
	CudaBuffer<value_type> &real_temp,
	CudaBuffer<value_type> &real_temp2,
	CalculationParameters &params,
	batchfftHandle plan, bool energy)
{
	dim3 grid, block;
	createKernelParams(block, grid, state.len(), MAX_THREADS_NUM);

	cufftSafeCall(batchfftExecute(plan, (hipfftComplex*)state, (hipfftComplex*)complex_temp, HIPFFT_BACKWARD));
	multiply<<<grid, block>>>(complex_temp, 1.0 / state.len());
	cutilCheckMsg("multiply");

	if(energy)
	{
		calculateNonlinearEnergyPart<<<grid, block>>>(real_temp, state);
		cutilCheckMsg("calculateNonlinearEnergyPart");
	}
	else
	{
		calculateNonlinearMuPart<<<grid, block>>>(real_temp, state);
		cutilCheckMsg("calculateNonlinearMuPart");
	}

	combineNonlinearAndDifferential<<<grid, block>>>(real_temp, state, complex_temp);
	cutilCheckMsg("combineNonlinearAndDifferential");

	return reduce<value_type>(real_temp, real_temp2, state.len(), 1) * params.dx * params.dy * params.dz;
}

// returns number if particles for given state
// *_temp values are spoiled after call
value_type calculateParticles(CudaBuffer<value_pair> &state, CudaBuffer<value_type> &real_temp,
	CudaBuffer<value_type> &real_temp2, CalculationParameters &params)
{
	dim3 grid, block;
	createKernelParams(block, grid, state.len(), MAX_THREADS_NUM);

	calculateModules<<<grid, block>>>(real_temp, state);
	cutilCheckMsg("calculateModules");

	return reduce<value_type>(real_temp, real_temp2, state.len(), 1) *
		params.dx * params.dy * params.dz;
}

void calculateSteadyState(value_pair *h_steady_state, CalculationParameters &params)
{
	batchfftHandle plan;
	dim3 block, grid;
	value_type E = 0;

	CudaBuffer<value_pair> a(params.cells), complex_temp(params.cells);
	CudaBuffer<value_type> real_temp(params.cells), real_temp2(params.cells);

	createKernelParams(block, grid, params.cells, MAX_THREADS_NUM);
	cufftSafeCall(batchfftPlan3d(&plan, params.nvz, params.nvy, params.nvx, HIPFFT_C2C, 1));

	//initial GP solution in k-space
	fillWithTFGroundState<<<grid, block>>>(a);
	cutilCheckMsg("fillWithTFGroundState");

	// normalize initial conditions
	value_type N = calculateParticles(a, real_temp, real_temp2, params);
	printf("N = %f\n", N);
	multiply<<<grid, block>>>(a, sqrt(params.N / N));

	// FFT into k-space
	cufftSafeCall(batchfftExecute(plan, (hipfftComplex*)a, (hipfftComplex*)a, HIPFFT_BACKWARD));
	multiply<<<grid, block>>>(a, 1.0 / params.cells);
	cutilCheckMsg("multiply");

	//////////////////////////////////////////////////////////////////////////
	// 	Starts GP loop in time: calculate mean-field steady-state
	//////////////////////////////////////////////////////////////////////////
	value_type t = 0;
	while(1)
	{
		// Linear propagate in k-space
		propagateKSpaceImaginaryTime<<<grid, block>>>(a);
		cutilCheckMsg("propagateKSpaceImaginaryTime");

		// FFT into x-space
		cufftSafeCall(batchfftExecute(plan, (hipfftComplex*)a, (hipfftComplex*)a, HIPFFT_FORWARD));

		propagateXSpaceOneComponent<<<grid, block>>>(a);
		cutilCheckMsg("propagateToEndpoint");

		// FFT into k-space
		cufftSafeCall(batchfftExecute(plan, (hipfftComplex*)a, (hipfftComplex*)a, HIPFFT_BACKWARD));
		multiply<<<grid, block>>>(a, 1.0 / params.cells);
		cutilCheckMsg("multiply");

		// Linear propagate in k-space
		propagateKSpaceImaginaryTime<<<grid, block>>>(a);
		cutilCheckMsg("propagateKSpaceImaginaryTime");

		// Renormalization

		// FFT into x-space
		cufftSafeCall(batchfftExecute(plan, (hipfftComplex*)a, (hipfftComplex*)a, HIPFFT_FORWARD));

		// Normalize
		N = calculateParticles(a, real_temp, real_temp2, params);
		//printf("N = %f\n", N);
		multiply<<<grid, block>>>(a, sqrt(params.N / N));

		// Calculate energy
		value_type new_E = calculateStateIntegral(a, complex_temp, real_temp, real_temp2, params, plan, true);

		// FFT into k-space
		cufftSafeCall(batchfftExecute(plan, (hipfftComplex*)a, (hipfftComplex*)a, HIPFFT_BACKWARD));
		multiply<<<grid, block>>>(a, 1.0 / params.cells);
		cutilCheckMsg("multiply");

		if(abs((new_E - E) / new_E) < 0.000001)
			break;

		E = new_E;
		t += params.dt_steady;
	} //end time loop

	//FFT into x-space
	cufftSafeCall(batchfftExecute(plan, (hipfftComplex*)a, (hipfftComplex*)a, HIPFFT_FORWARD));

	// save steady state
	a.copyTo(h_steady_state);

	E = calculateStateIntegral(a, complex_temp, real_temp, real_temp2, params, plan, true);
	printf("E = %f\n", E / params.N);

	value_type mu = calculateStateIntegral(a, complex_temp, real_temp, real_temp2, params, plan, false);
	printf("mu = %f\n", mu / params.N);

	batchfftDestroy(plan);
}

// Propagate k-state for evolution calculation
void propagate(CalculationParameters &params, EvolutionState &state, value_type dt)
{
	propagateKSpaceRealTime<<<state.grid, state.block>>>(state.a, state.b, dt);
	cutilCheckMsg("propagateKSpaceRealTime");

	//FFT into x-space
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.a, (hipfftComplex*)state.a, HIPFFT_FORWARD));
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.b, (hipfftComplex*)state.b, HIPFFT_FORWARD));

	propagateXSpaceTwoComponent<<<state.grid, state.block>>>(state.a, state.b, dt);
	cutilCheckMsg("propagateMidpoint");

	//FFT into k-space
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.a, (hipfftComplex*)state.a, HIPFFT_BACKWARD));
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.b, (hipfftComplex*)state.b, HIPFFT_BACKWARD));
	multiplyPair<<<state.grid, state.block>>>(state.a, state.b, 1.0 / params.cells);
	cutilCheckMsg("multiplyPair");

	//Linear propagate a,b-field
	propagateKSpaceRealTime<<<state.grid, state.block>>>(state.a, state.b, dt);
	cutilCheckMsg("propagateKSpaceRealTime");
}

// initialize evolution state
void initEvolution(value_pair *h_steady_state, CalculationParameters &params, EvolutionState &state)
{
	int size = params.cells * params.ne;
	CudaBuffer<value_pair> noise(size * 2), steady_state(params.cells);

	steady_state.copyFrom(h_steady_state);

	// initialize ensembles
	srand(time(0));
	fillWithNormalDistribution(noise, 0.5f);
	initializeEnsembles<<<state.grid, state.block>>>(state.a, state.b, steady_state, noise);
	cutilCheckMsg("initializeEnsembles");

	// FFT into k-space
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.a, (hipfftComplex*)state.a, HIPFFT_BACKWARD));
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.b, (hipfftComplex*)state.b, HIPFFT_BACKWARD));
	multiplyPair<<<state.grid, state.block>>>(state.a, state.b, 1.0 / params.cells);
	cutilCheckMsg("multiplyPair");

	// Equilibration phase
	for(value_type t = 0; t <= params.t_equilib; t += params.dt_evo)
		propagate(params, state, params.dt_evo);

	applyHalfPiPulse<<<state.grid, state.block>>>(state.a, state.b);
	cutilCheckMsg("applyBraggPulse");
}

// reduce sparse elements instead of neighbouring ones
// data in *in is spoiled after call
template<class T>
void sparseReduce(CudaBuffer<T> &out, CudaBuffer<T> &in, int length, int final_length = 1)
{
	int coeff = length / final_length;

	if(coeff == 1)
	{
		out.copyFrom(in);
		return;
	}

	// transpose cannot handle matrices with dimensions less than 16
	if(coeff >= 16)
	{
		cutilSafeCall(transpose<T>(out, in, final_length, coeff, 1));
		reduce<T>(out, in, length, final_length);
	}
	else
	{
		dim3 block, grid;
		createKernelParams(block, grid, final_length, MAX_THREADS_NUM);
		smallReduce<T><<<grid, block>>>(out, in, coeff);
		cutilCheckMsg("smallReduce");
	}
}

value_type getComponentRatio(CalculationParameters &params, EvolutionState &state, value_type angle)
{
	halfPiRotate<<<state.grid, state.block>>>(state.dens_a, state.dens_b, state.a, state.b, angle);
	cutilCheckMsg("halfPiRotate");

	value_type N1 = reduce<value_type>(state.dens_a, state.temp, params.cells * params.ne, 1);
	value_type N2 = reduce<value_type>(state.dens_b, state.temp, params.cells * params.ne, 1);
	return (N1 - N2) / (N1 + N2);
}

value_type getVisibility(CalculationParameters &params, EvolutionState &state)
{
	value_type max = 0;
	for(value_type alpha = 0; alpha < 2 * M_PI; alpha += 0.5)
	{
		value_type ratio = getComponentRatio(params, state, alpha);
		if(abs(ratio) > max)
			max = abs(ratio);
	}
	return max;
}

void printComponentRatioAxialProjection(CalculationParameters &params, EvolutionState &state)
{
	halfPiRotate<<<state.grid, state.block>>>(state.dens_a, state.dens_b, state.a, state.b, 0);
	cutilCheckMsg("halfPiRotate");

	sparseReduce(state.temp, state.dens_a, params.cells * params.ne, params.cells);
	reduce<value_type>(state.temp, state.dens_a, params.cells, params.nvz);

	sparseReduce(state.temp2, state.dens_b, params.cells * params.ne, params.cells);
	reduce<value_type>(state.temp2, state.dens_b, params.cells, params.nvz);

	value_type *a_proj = new value_type[params.nvz];
	value_type *b_proj = new value_type[params.nvz];

	state.temp.copyTo(a_proj, params.nvz);
	state.temp2.copyTo(b_proj, params.nvz);

	printf("%f", state.t * params.t_rho * 1000);
	for(int i = 0; i < params.nvz; i++)
		printf(" %f", (a_proj[i] - b_proj[i]) / (a_proj[i] + b_proj[i]));
	printf("\n");

	delete[] a_proj;
	delete[] b_proj;
}

void calculateAverages(CalculationParameters &params, EvolutionState &state)
{
	// calculate norm (sum (|psi|^2) = N)
	calculateModules<<<state.grid, state.block>>>(state.dens_a, state.a);
	cutilCheckMsg("calculateModules");
	value_type a_avg_module = reduce<value_type>(state.dens_a, state.temp, params.cells * params.ne, 1) /
		(params.cells * params.ne);

	calculateModules<<<state.grid, state.block>>>(state.dens_b, state.b);
	cutilCheckMsg("calculateModules");
	value_type b_avg_module = reduce<value_type>(state.dens_b, state.temp, params.cells * params.ne, 1) /
		(params.cells * params.ne);

	printf("%f\n", (a_avg_module + b_avg_module) * params.cells * params.dx * params.dy * params.dz);
	value_type norm = params.N / ((a_avg_module + b_avg_module) * params.cells);

	// Calculate averages
	cutilSafeCall(hipMemcpy(state.complex_t1, state.a, params.cells * params.ne * sizeof(value_pair), hipMemcpyDeviceToDevice));
	sparseReduce<value_pair>(state.complex_t2, state.complex_t1, params.cells * params.ne, params.cells);
	kernelAverage<<<state.grid, state.block>>>(state.complex_t1, state.complex_t2, state.a);
	value_pair a_avg = reduce<value_pair>(state.complex_t1, state.complex_t2, params.cells * params.ne, 1) /
		(params.cells * params.ne);

	cutilSafeCall(hipMemcpy(state.complex_t1, state.b, params.cells * params.ne * sizeof(value_pair), hipMemcpyDeviceToDevice));
	sparseReduce<value_pair>(state.complex_t2, state.complex_t1, params.cells * params.ne, params.cells);
	kernelAverage<<<state.grid, state.block>>>(state.complex_t1, state.complex_t2, state.b);
	value_pair b_avg = reduce<value_pair>(state.complex_t1, state.complex_t2, params.cells * params.ne, 1) /
		(params.cells * params.ne);

	printf("Avgs: Re(a): %f, Im(a): %f, Re(b): %f, Im(b): %f\n",
	       a_avg.x * norm, a_avg.y * norm, b_avg.x * norm, b_avg.y * norm);
//	printf("Avgs: Re = %f, Im = %f\n", (a_re_avg + b_re_avg) * norm, (a_im_avg + b_im_avg) * norm);
}

// propagate system and fill current state graph data
void calculateEvolution(CalculationParameters &params, EvolutionState &state, value_type dt)
{
	propagate(params, state, dt);
	state.t += dt;

	//FFT into x-space
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.a, (hipfftComplex*)state.a, HIPFFT_FORWARD));
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.b, (hipfftComplex*)state.b, HIPFFT_FORWARD));
	cutilSafeCall(hipDeviceSynchronize());
/*
	if(!state.pi_pulse_applied && state.t * params.t_rho >= 0.03)
	{
		state.pi_pulse_applied = true;
		applyPiPulse<<<state.grid, state.block>>>(state.a, state.b);
		cutilCheckMsg("applyPiPulse");
	}
 */
//	printf("%f %f\n", state.t * params.t_rho * 1000, getVisibility(params, state));
//	printf("%f %f\n", state.t * params.t_rho * 1000, getComponentRatio(params, state, 0));
	printComponentRatioAxialProjection(params, state);

	// second pi/2 pulse
	halfPiRotate<<<state.grid, state.block>>>(state.dens_a, state.dens_b, state.a, state.b, 0);
	cutilCheckMsg("halfPiRotate");

	//FFT into k-space
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.a, (hipfftComplex*)state.a, HIPFFT_BACKWARD));
	cufftSafeCall(batchfftExecute(state.plan, (hipfftComplex*)state.b, (hipfftComplex*)state.b, HIPFFT_BACKWARD));

	cutilSafeCall(hipDeviceSynchronize());

	multiplyPair<<<state.grid, state.block>>>(state.a, state.b, 1.0 / params.cells);
	cutilCheckMsg("multiplyPair");

	// reduce<value_type>() reduces neighbouring values first, and we need to reduce
	// values for each particle separately
	// so we transform [ensemble1, ensemble2, ...] to [particle1, particle2, ...] using transpose<value_type>()
	// and then perform reduce<value_type>()
	sparseReduce(state.temp, state.dens_a, params.cells * params.ne, params.cells);

	// projection on XY plane
	cutilSafeCall(transpose<value_type>(state.temp2, state.temp, params.nvx * params.nvy, params.nvz, 1));
	reduce<value_type>(state.temp2, state.dens_a, params.cells, params.nvx * params.nvy);
	state.dens_a_xy.copyFrom(state.temp2, params.nvx * params.nvy);

	// projection on YZ plane
	reduce<value_type>(state.temp, state.temp2, params.cells, params.nvy * params.nvz);
	state.dens_a_zy.copyFrom(state.temp2, params.nvy * params.nvz);
	cutilSafeCall(transpose<value_type>(state.dens_a_zy, state.temp, params.nvy, params.nvz, 1));

	sparseReduce(state.temp, state.dens_b, params.cells * params.ne, params.cells);

	// projection on XY plane
	cutilSafeCall(transpose<value_type>(state.temp2, state.temp, params.nvx * params.nvy, params.nvz, 1));
	reduce<value_type>(state.temp2, state.dens_b, params.cells, params.nvx * params.nvy);
	state.dens_b_xy.copyFrom(state.temp2, params.nvx * params.nvy);

	// projection on YZ plane
	reduce<value_type>(state.temp, state.temp2, params.cells, params.nvy * params.nvz);
	state.dens_b_zy.copyFrom(state.temp2, params.nvy * params.nvz);
	cutilSafeCall(transpose<value_type>(state.dens_b_zy, state.temp, params.nvy, params.nvz, 1));
}

// Draw graphs from current state to provided buffers
void drawState(CalculationParameters &params, EvolutionState &state, CudaTexture &a_xy_tex,
	CudaTexture &b_xy_tex, CudaTexture &a_zy_tex, CudaTexture &b_zy_tex)
{

	value_type scale = 3.0 * params.N * params.ne / (params.cells * params.dx * params.dy * params.dz);
	value_type xy_scale = scale * params.nvz;
	value_type zy_scale = scale * params.nvx;

	drawData(a_xy_tex, state.dens_a_xy, xy_scale);
	drawData(b_xy_tex, state.dens_b_xy, xy_scale);
	drawData(a_zy_tex, state.dens_a_zy, zy_scale);
	drawData(b_zy_tex, state.dens_b_zy, zy_scale);

	float4 *b_zy_buf = b_zy_tex.map();
	hipMemcpy(state.to_bmp, b_zy_buf, params.nvz * params.nvy * sizeof(float4), hipMemcpyDeviceToHost);
	b_zy_tex.unmap();
}
